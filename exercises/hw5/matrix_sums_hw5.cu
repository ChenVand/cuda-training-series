
#include <hip/hip_runtime.h>
#include <stdio.h>

// error checking macro
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)


const size_t DSIZE = 16384;      // matrix side dimension
const int block_size = 256;  // CUDA maximum is 1024
// matrix row-sum kernel
__global__ void row_sums(const float *A, float *sums, size_t ds){

  int idx = threadIdx.x + blockIdx.x*blockDim.x; // create typical 1D thread index from built-in variables
  if (idx < ds){
    float sum = 0.0f;
    for (size_t i = 0; i < ds; i++)
      sum += A[idx*ds + i];         // write a for loop that will cause the thread to iterate across a row, keeeping a running sum, and write the result to sums
    sums[idx] = sum;
  }
}

__global__ void row_sums_new(const float *A, float *sums, size_t ds){

  // __shared__ float partial_sum[32];
  int tid = threadIdx.x;
  int lane = tid % warpSize;
  int warpID = tid / warpSize;
  unsigned mask = 0xFFFFFFFFU;
  float val;
  size_t row;
  for (size_t row_batch = 0; row_batch < DSIZE; row_batch+=gridDim.x){
    // if (tid < 32) partial_sum[tid]=0.0f; //A little out of place but should be in this loop
    row = row_batch + blockIdx.x;
    if (row >= DSIZE) return;
    for (size_t col_batch = 0; col_batch < DSIZE; col_batch+=blockDim.x){
      if (col_batch + tid < DSIZE)
        val = A[row*DSIZE + col_batch + tid];
      else val = 0;

      for (int offset = warpSize/2; offset > 0; offset >>= 1) 
         val += __shfl_down_sync(mask, val, offset);

      if  (lane == 0) atomicAdd(&sums[row], val);
      // if  (lane == 0) partial_sum[warpID] += val;
    }

    // __syncthreads();
    // if (warpID==0){
    //   val = partial_sum[tid];
    //   for (int offset = warpSize/2; offset > 0; offset >>= 1) 
    //      val += __shfl_down_sync(mask, val, offset);

    //   if  (tid == 0) sums[row] = val;
    // }
  }
}

// matrix column-sum kernel
__global__ void column_sums(const float *A, float *sums, size_t ds){

  int idx = threadIdx.x+blockDim.x*blockIdx.x; // create typical 1D thread index from built-in variables
  if (idx < ds){
    float sum = 0.0f;
    for (size_t i = 0; i < ds; i++)
      sum += A[idx+ds*i];         // write a for loop that will cause the thread to iterate down a column, keeeping a running sum, and write the result to sums
    sums[idx] = sum;
}}
bool validate(float *data, size_t sz){
  for (size_t i = 0; i < sz; i++)
    if (data[i] != (float)sz) {printf("results mismatch at %lu, was: %f, should be: %f\n", i, data[i], (float)sz); return false;}
    return true;
}
int main(){

  float *h_A, *h_sums, *d_A, *d_sums;
  h_A = new float[DSIZE*DSIZE];  // allocate space for data in host memory
  h_sums = new float[DSIZE]();
  for (int i = 0; i < DSIZE*DSIZE; i++)  // initialize matrix in host memory
    h_A[i] = 1.0f;
  hipMalloc(&d_A, DSIZE*DSIZE*sizeof(float));  // allocate device space for A
  hipMalloc(&d_sums, DSIZE*sizeof(float));  // allocate device space for vector d_sums
  cudaCheckErrors("hipMalloc failure"); // error checking
  // copy matrix A to device:
  hipMemcpy(d_A, h_A, DSIZE*DSIZE*sizeof(float), hipMemcpyHostToDevice);
  cudaCheckErrors("hipMemcpy H2D failure");
  //cuda processing sequence step 1 is complete
  row_sums<<<(DSIZE+block_size-1)/block_size, block_size>>>(d_A, d_sums, DSIZE);
  cudaCheckErrors("kernel launch failure");
  //cuda processing sequence step 2 is complete
  // copy vector sums from device to host:
  hipMemcpy(h_sums, d_sums, DSIZE*sizeof(float), hipMemcpyDeviceToHost);
  //cuda processing sequence step 3 is complete
  cudaCheckErrors("kernel execution failure or hipMemcpy H2D failure");
  if (!validate(h_sums, DSIZE)) return -1; 
  printf("row sums correct!\n");
  hipMemset(d_sums, 0, DSIZE*sizeof(float));

  row_sums_new<<<(DSIZE+block_size-1)/block_size, block_size>>>(d_A, d_sums, DSIZE);
  cudaCheckErrors("kernel launch failure");
  //cuda processing sequence step 2 is complete
  // copy vector sums from device to host:
  hipMemcpy(h_sums, d_sums, DSIZE*sizeof(float), hipMemcpyDeviceToHost);
  //cuda processing sequence step 3 is complete
  cudaCheckErrors("kernel execution failure or hipMemcpy H2D failure");
  if (!validate(h_sums, DSIZE)) return -1; 
  printf("row sums new correct!\n");
  hipMemset(d_sums, 0, DSIZE*sizeof(float));

  column_sums<<<(DSIZE+block_size-1)/block_size, block_size>>>(d_A, d_sums, DSIZE);
  cudaCheckErrors("kernel launch failure");
  //cuda processing sequence step 2 is complete
  // copy vector sums from device to host:
  hipMemcpy(h_sums, d_sums, DSIZE*sizeof(float), hipMemcpyDeviceToHost);
  //cuda processing sequence step 3 is complete
  cudaCheckErrors("kernel execution failure or hipMemcpy H2D failure");
  if (!validate(h_sums, DSIZE)) return -1; 
  printf("column sums correct!\n");
  return 0;
}
  
